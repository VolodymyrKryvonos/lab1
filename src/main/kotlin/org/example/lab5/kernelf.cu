#include "hip/hip_runtime.h"
 extern "C"
__global__ void multiply(int *n, float *matrix, float *multiplayer)
{
    for(int i = 0; i<n;i++){
        for(int j = 0; j<n;j++){
            float sum=0;
            for(int k = 0; k<n; k++){
                sum+= matrix[i*n+j] * multiplayer[i*n+k]
            }
            matrix[i*n+j]=sum
        }
    }
}